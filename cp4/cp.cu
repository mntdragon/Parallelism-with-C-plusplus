#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/for_each.h>
#include <thrust/execution_policy.h>
#include <thrust/transform.h>

// CUDA error checking
static inline void cudaErrorPrint(hipError_t err, const char *context)
{
    if (err != hipSuccess)
    {
        std::cerr << "CUDA error: " << context << ": "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define ERRORVERIFIER(x) cudaErrorPrint(x, #x)

// Calculate the smallest integer greater than or equal to the result of a division
static inline int divup(int a, int b)
{
    return (a + b - 1) / b;
}

// Round up a number to the nearest multiple of another number
static inline int roundup(int a, int b)
{
    return divup(a, b) * b;
}

// Device function to compute squared difference
__device__ float squared_diff(float x, float mean)
{
    return (x - mean) * (x - mean);
}

// Device function to normalize data
__device__ float normalize(float x, float mean, float resultMatrix)
{
    return (x - mean) / resultMatrix;
}

// Normalize the matrix to ensure that the correlation values are between -1 and 1
__global__ void normalizeMatrixKernel(int ny, int nx, float *data, float *ntdata)
{
    int y = blockIdx.x;
    if (y >= ny)
        return;

    float sum = thrust::transform_reduce(thrust::seq, data + y * nx, data + (y + 1) * nx, thrust::identity<float>(), 0.0f, thrust::plus<float>());
    float mean = sum / static_cast<float>(nx);

    float rs = thrust::transform_reduce(thrust::seq, data + y * nx, data + (y + 1) * nx, thrust::identity<float>(), 0.0f, thrust::plus<float>(), [=] __device__(float x) { return squared_diff(x, mean); });
    float resultMatrix = sqrt(rs);

    for (int x = 0; x < nx; ++x)
    {
        ntdata[y + x * ny] = normalize(data[x + y * nx], mean, resultMatrix);
    }
}

// Perform the matrix multiplication to calculate the correlation matrix
__global__ void multiplyMatricesKernel(int ny, int nx, float *ntdata, float *resultMatrix)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= ny || j >= ny)
        return;

    float sum = thrust::inner_product(thrust::seq, ntdata + j * nx, ntdata + (j + 1) * nx, ntdata + i * nx, 0.0f, thrust::plus<float>(), thrust::multiplies<float>());
    resultMatrix[j + i * ny] = sum;
}

void correlate(int ny, int nx, const float *data, float *result)
{
    const int THREAD_BLOCK_SIZE = 16;
    int rn = ny * ny;

    // Allocate device vectors using thrust
    thrust::device_vector<float> deviceInputData(data, data + ny * nx);
    thrust::device_vector<float> deviceNormalizedData(ny * nx);
    thrust::device_vector<float> deviceResultMatrix(rn);

    hipStream_t stream1, stream2;
    ERRORVERIFIER(hipStreamCreate(&stream1));
    ERRORVERIFIER(hipStreamCreate(&stream2));

    // Normalize data before performing normalization operations
    normalizeMatrixKernel<<<ny, 1, 0, stream1>>>(ny, nx, thrust::raw_pointer_cast(deviceInputData.data()), thrust::raw_pointer_cast(deviceNormalizedData.data()));

    // Multiply matrices
    dim3 threadBlock(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);
    dim3 gridDim(divup(ny, threadBlock.x), divup(ny, threadBlock.y));
    multiplyMatricesKernel<<<gridDim, threadBlock, 0, stream2>>>(ny, nx, thrust::raw_pointer_cast(deviceNormalizedData.data()), thrust::raw_pointer_cast(deviceResultMatrix.data()));

    // Copy result back to host
    thrust::copy(deviceResultMatrix.begin(), deviceResultMatrix.end(), result);

    ERRORVERIFIER(hipStreamSynchronize(stream2));
    ERRORVERIFIER(hipStreamDestroy(stream1));
    ERRORVERIFIER(hipStreamDestroy(stream2));
}
