#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <cmath>

// Utility function to divide and round up
static inline int divup(int a, int b) {
    return (a + b - 1) / b;
}

// Utility function to round up to the nearest multiple of b
static inline int roundup(int a, int b) {
    return divup(a, b) * b;
}

// Utility function to handle CUDA errors
static inline void cudaErrorPrint(hipError_t err, const char* context) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error: " << context << " -> Due to "
                  << hipGetErrorString(err) << std::endl;
        std::exit(EXIT_FAILURE);
    }
}

#define ERRORHANDLER(x) cudaErrorPrint(x, #x)

/**
 * Kernel function for normalizing rows of a matrix.
 *
 * @param numRows Number of rows in the matrix.
 * @param numCols Number of columns in the matrix.
 * @param inputData Pointer to the input data matrix (device memory).
 * @param normalizedData Pointer to the normalized data matrix (device memory).
 */
__global__ void normalizationKernel(int numRows, int numCols, float* inputData, float* normalizedData) {
    int row = blockIdx.x;
    if (row >= numRows) return;

    // Calculate the mean of the row
    float sum = 0.0f;
    for (int col = 0; col < numCols; ++col) {
        sum += inputData[col + row * numCols];
    }
    float mean = sum / static_cast<float>(numCols);

    // Calculate the standard deviation of the row
    float sumSquaredDiffs = 0.0f;
    for (int col = 0; col < numCols; ++col) {
        float value = inputData[col + row * numCols];
        sumSquaredDiffs += (value - mean) * (value - mean);
    }
    float stddev = sqrt(sumSquaredDiffs);

    // Normalize the row
    for (int col = 0; col < numCols; ++col) {
        float value = inputData[col + row * numCols];
        normalizedData[row + col * numRows] = (value - mean) / stddev;
    }
}

/**
 * Kernel function for performing matrix multiplication.
 *
 * @param numRows Number of rows in the matrices.
 * @param numCols Number of columns in the input matrices.
 * @param normalizedData Pointer to the normalized data matrix (device memory).
 * @param result Pointer to the result matrix (device memory).
 */
__global__ void matmulKernel(int numRows, int numCols, float* normalizedData, float* result) {
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    int col = threadIdx.y + blockIdx.y * blockDim.y;
    if (row >= numRows || col >= numRows) return;

    float sum = 0.0f;
    if (row <= col) {
        for (int k = 0; k < numCols; ++k) {
            float x = normalizedData[numRows * k + col];
            float y = normalizedData[numRows * k + row];
            sum += x * y;
        }
    }
    result[col + row * numRows] = sum;
}

/**
 * Function to correlate rows of a matrix using CUDA.
 *
 * @param numRows Number of rows in the input matrix.
 * @param numCols Number of columns in the input matrix.
 * @param data Pointer to the input matrix (host memory).
 * @param result Pointer to the result matrix (host memory).
 */
void correlate(int numRows, int numCols, const float* data, float* result) {
    int dataSize = numRows * numCols * sizeof(float);
    int resultSize = numRows * numRows * sizeof(float);

    float* dInputData = nullptr;
    ERRORHANDLER(hipMalloc((void**)&dInputData, dataSize));

    float* dNormalizedData = nullptr;
    ERRORHANDLER(hipMalloc((void**)&dNormalizedData, dataSize));

    float* dResult = nullptr;
    ERRORHANDLER(hipMalloc((void**)&dResult, resultSize));

    ERRORHANDLER(hipMemcpy(dInputData, data, dataSize, hipMemcpyHostToDevice));

    int nBlocks = roundup(numRows, 64);
    normalizationKernel<<<nBlocks, 1>>>(numRows, numCols, dInputData, dNormalizedData);
    ERRORHANDLER(hipDeviceSynchronize());
    ERRORHANDLER(hipGetLastError());

    dim3 dimBlock(16, 16);
    dim3 dimGrid(divup(numRows, dimBlock.x), divup(numRows, dimBlock.y));
    matmulKernel<<<dimGrid, dimBlock>>>(numRows, numCols, dNormalizedData, dResult);
    ERRORHANDLER(hipGetLastError());

    ERRORHANDLER(hipMemcpy(result, dResult, resultSize, hipMemcpyDeviceToHost));
    ERRORHANDLER(hipFree(dInputData));
    ERRORHANDLER(hipFree(dNormalizedData));
    ERRORHANDLER(hipFree(dResult));
}
